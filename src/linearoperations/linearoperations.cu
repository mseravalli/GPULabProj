#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    linearoperations.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * linearoperations.cu
 *
 *  Created on: Aug 3, 2012
 *      Author: steinbrf
 */


#include <auxiliary/cuda_basic.cuh>

hipChannelFormatDesc linearoperation_float_tex = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_linearoperation;
bool linearoperation_textures_initialized = false;


#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE   21    // maximum allowed kernel radius + 1
__constant__ float constKernel[MAXKERNELSIZE];


void setTexturesLinearOperations(int mode){
	tex_linearoperation.addressMode[0] = hipAddressModeClamp;
	tex_linearoperation.addressMode[1] = hipAddressModeClamp;
	if(mode == 0)tex_linearoperation.filterMode = hipFilterModePoint;
	else tex_linearoperation.filterMode = hipFilterModeLinear;
	tex_linearoperation.normalized = false;
}


#define LO_TEXTURE_OFFSET 0.5f
#define LO_RS_AREA_OFFSET 0.0f

#ifdef DGT400
#define LO_BW 32
#define LO_BH 16
#else
#define LO_BW 16
#define LO_BH 16
#endif


#ifndef RESAMPLE_EPSILON
#define RESAMPLE_EPSILON 0.005f
#endif

#ifndef atomicAdd
__device__ float atomicAdd(float* address, double val)
{
	unsigned int* address_as_ull = (unsigned int*)address;
	unsigned int old = *address_as_ull, assumed;
	do{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__float_as_int(val + __int_as_float(assumed)));
	}	while (assumed != old);
	return __int_as_float(old);
}

#endif


//NOTE global was added to the function signature
//TODO add texture!!
__global__ void backwardRegistrationBilinearValueTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		float value,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  // check if x is within the boundaries
  if (!(x < nx && y < ny)) {
    return;
  }

	float hx_1 = 1.0f/hx;
	float hy_1 = 1.0f/hy;
  float ii_fp = x+(flow1_g[y*pitchf1_in+x]*hx_1);
  float jj_fp = y+(flow2_g[y*pitchf1_in+x]*hy_1);
  
  if((ii_fp < 0.0f) || (jj_fp < 0.0f)
  			 || (ii_fp > (float)(nx-1)) || (jj_fp > (float)(ny-1))){
  	out_g[y*pitchf1_out+x] = value;
  }
  else if(!isfinite(ii_fp) || !isfinite(jj_fp)){
  	out_g[y*pitchf1_out+x] = value;
  }
  else{
  	int xx = (int)floor(ii_fp);
  	int yy = (int)floor(jj_fp);
  
  	int xx1 = xx == nx-1 ? xx : xx+1;
  	int yy1 = yy == ny-1 ? yy : yy+1;
  
  	float xx_rest = ii_fp - (float)xx;
  	float yy_rest = jj_fp - (float)yy;
  
  	out_g[y*pitchf1_out+x] = (1.0f-xx_rest)*(1.0f-yy_rest) * in_g[yy   * pitchf1_in + xx]
  		                     + xx_rest*(1.0f-yy_rest)        * in_g[yy   * pitchf1_in + xx1]
  		                     + (1.0f-xx_rest)*yy_rest        * in_g[(yy1)* pitchf1_in + xx]
  		                     + xx_rest * yy_rest             * in_g[(yy1)* pitchf1_in + xx1];
  }

}

//NOTE global was added to the function signature
__global__ void backwardRegistrationBilinearFunctionGlobal
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  // check if x is within the boundaries
  if (!(x < nx && y < ny)) {
    return;
  }
  
  const float xx = (float)x+flow1_g[y*pitchf1_in+x]/hx;
  const float yy = (float)y+flow2_g[y*pitchf1_in+x]/hy;
  
  int xxFloor = (int)floor(xx);
  int yyFloor = (int)floor(yy);
  
  int xxCeil = xxFloor == nx-1 ? xxFloor : xxFloor+1;
  int yyCeil = yyFloor == ny-1 ? yyFloor : yyFloor+1;
  
  float xxRest = xx - (float)xxFloor;
  float yyRest = yy - (float)yyFloor;

  out_g[y*pitchf1_out+x] =
      (xx < 0.0f || yy < 0.0f || xx > (float)(nx-1) || yy > (float)(ny-1))
      ? constant_g[y*pitchf1_in+x] :
    (1.0f-xxRest)*(1.0f-yyRest) * in_g[yyFloor * pitchf1_in + xxFloor]
        + xxRest*(1.0f-yyRest)  * in_g[yyFloor * pitchf1_in + xxCeil]
        + (1.0f-xxRest)*yyRest  * in_g[yyCeil  * pitchf1_in + xxFloor]
        + xxRest * yyRest       * in_g[yyCeil  * pitchf1_in + xxCeil];
}

void backwardRegistrationBilinearFunctionTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me, if you want ###
}




void forewardRegistrationBilinearAtomic
(
		const float *flow1_g,
		const float *flow2_g,
		const float *in_g,
	  float       *out_g,
		int         nx,
		int         ny,
		int         pitchf1
)
{
	// ### Implement me ###
}




void gaussBlurSeparateMirrorGpu
(
		float *in_g,
		float *out_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float sigmax,
		float sigmay,
		int   radius,
		float *temp_g,
		float *mask
)
{
	// ### Implement me ###
}



//NOTE global was added to the function signature
__global__ void resampleAreaParallelSeparate
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  // check if x is within the boundaries
  if (!(x < nx_in && y < ny_in)) {
    return;
  }

  float hx = (float)(nx_in)/(float)(nx_out);
  float hy = (float)(ny_in)/(float)(ny_out);

  int p = y*pitchf1_out + x;

  // resampling in x
	float px = (float)x * hx;
	float left = ceil(px) - px;
	if(left > hx) left = hx;
	float midx = hx - left;
	float right = midx - floorf(midx);
	midx = midx - right;
  int nx_orig = pitchf1_in;

	help_g[p] = 0.0f;

	if(left > 0.0f){
		help_g[p] += in_g[y*nx_orig+(int)(floor(px))]*left*scalefactor;
		px+= 1.0f;
	}
	while(midx > 0.0f){
		help_g[p] += in_g[y*nx_orig+(int)(floor(px))]*scalefactor;
		px += 1.0f;
		midx -= 1.0f;
	}
	if(right > RESAMPLE_EPSILON)	{
		help_g[p] += in_g[y*nx_orig+(int)(floor(px))]*right*scalefactor;
	}

  // resampling in y
  float py = (float)y * hy;
  float top = ceil(py) - py;
  if(top > hy) top = hy;
  float midy = hy - top;
  float bottom = midy - floorf(midy);
  midy = midy - bottom;
  
  out_g[p] = 0.0f;
  
  if(top > 0.0f){
  	out_g[p] += help_g[(int)(floor(py))*pitchf1_out+x]*top*scalefactor;
  	py += 1.0f;
  }
  while(midy > 0.0f){
  	out_g[p] += help_g[(int)(floor(py))*pitchf1_out+x]*scalefactor;
  	py += 1.0f;
  	midy -= 1.0f;
  }
  if(bottom > RESAMPLE_EPSILON){
  	out_g[p] += help_g[(int)(floor(py))*pitchf1_out+x]*bottom*scalefactor;
  }

}

void resampleAreaParallelSeparateAdjoined
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
}


__global__ void addKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y*pitchf1 + x;
	if (x < nx && y < ny) {
		accumulator_g[idx] += increment_g[idx];
	}
}

__global__ void subKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y*pitchf1 + x;
	if (x < nx && y < ny) {
		accumulator_g[idx] -= increment_g[idx];
	}
}

__global__ void setKernel
(
		float *field_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float value
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y*pitchf1 + x;
	if (x < nx && y < ny) {
		field_g[idx] = value;
	}
}

