#include "hip/hip_runtime.h"
/****************************************************************************\
*      --- Practical Course: GPU Programming in Computer Vision ---
*
* time:    winter term 2012/13 / March 11-18, 2013
*
* project: superresolution
* file:    linearoperations.cu
*
*
* implement all functions with ### implement me ### in the function body
\****************************************************************************/

/*
 * linearoperations.cu
 *
 *  Created on: Aug 3, 2012
 *      Author: steinbrf
 */


#include <auxiliary/cuda_basic.cuh>

hipChannelFormatDesc linearoperation_float_tex = hipCreateChannelDesc<float>();
texture<float, 2, hipReadModeElementType> tex_linearoperation;
bool linearoperation_textures_initialized = false;


#define MAXKERNELRADIUS     20    // maximum allowed kernel radius
#define MAXKERNELSIZE   21    // maximum allowed kernel radius + 1
__constant__ float constKernel[MAXKERNELSIZE];


void setTexturesLinearOperations(int mode){
	tex_linearoperation.addressMode[0] = hipAddressModeClamp;
	tex_linearoperation.addressMode[1] = hipAddressModeClamp;
	if(mode == 0)tex_linearoperation.filterMode = hipFilterModePoint;
	else tex_linearoperation.filterMode = hipFilterModeLinear;
	tex_linearoperation.normalized = false;
}


#define LO_TEXTURE_OFFSET 0.5f
#define LO_RS_AREA_OFFSET 0.0f

#ifdef DGT400
#define LO_BW 32
#define LO_BH 16
#else
#define LO_BW 16
#define LO_BH 16
#endif


#ifndef RESAMPLE_EPSILON
#define RESAMPLE_EPSILON 0.005f
#endif

#ifndef atomicAdd
__device__ float atomicAdd(float* address, double val)
{
	unsigned int* address_as_ull = (unsigned int*)address;
	unsigned int old = *address_as_ull, assumed;
	do{
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
				__float_as_int(val + __int_as_float(assumed)));
	}	while (assumed != old);
	return __int_as_float(old);
}

#endif




void backwardRegistrationBilinearValueTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		float value,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me ###
}

//NOTE global was added to the function signature
__global__ void backwardRegistrationBilinearFunctionGlobal
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  // check if x is within the boundaries
  if (!(x < nx && y < ny)) {
    return;
  }
  
  const float xx = (float)x+flow1_g[y*pitchf1_in+x]/hx;
  const float yy = (float)y+flow2_g[y*pitchf1_in+x]/hy;
  
  int xxFloor = (int)floor(xx);
  int yyFloor = (int)floor(yy);
  
  int xxCeil = xxFloor == nx-1 ? xxFloor : xxFloor+1;
  int yyCeil = yyFloor == ny-1 ? yyFloor : yyFloor+1;
  
  float xxRest = xx - (float)xxFloor;
  float yyRest = yy - (float)yyFloor;

  out_g[y*pitchf1_out+x] =
      (xx < 0.0f || yy < 0.0f || xx > (float)(nx-1) || yy > (float)(ny-1))
      ? constant_g[y*pitchf1_in+x] :
    (1.0f-xxRest)*(1.0f-yyRest) * in_g[yyFloor * pitchf1_in + xxFloor]
        + xxRest*(1.0f-yyRest)  * in_g[yyFloor * pitchf1_in + xxCeil]
        + (1.0f-xxRest)*yyRest  * in_g[yyCeil  * pitchf1_in + xxFloor]
        + xxRest * yyRest       * in_g[yyCeil  * pitchf1_in + xxCeil];
}

void backwardRegistrationBilinearFunctionTex
(
		const float *in_g,
		const float *flow1_g,
		const float *flow2_g,
		float *out_g,
		const float *constant_g,
		int   nx,
		int   ny,
		int   pitchf1_in,
		int   pitchf1_out,
		float hx,
		float hy
)
{
	// ### Implement me, if you want ###
}




void forewardRegistrationBilinearAtomic
(
		const float *flow1_g,
		const float *flow2_g,
		const float *in_g,
	  float       *out_g,
		int         nx,
		int         ny,
		int         pitchf1
)
{
	// ### Implement me ###
}




void gaussBlurSeparateMirrorGpu
(
		float *in_g,
		float *out_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float sigmax,
		float sigmay,
		int   radius,
		float *temp_g,
		float *mask
)
{
	// ### Implement me ###
}





void resampleAreaParallelSeparate
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
}

void resampleAreaParallelSeparateAdjoined
(
		const float *in_g,
		float *out_g,
		int   nx_in,
		int   ny_in,
		int   pitchf1_in,
		int   nx_out,
		int   ny_out,
		int   pitchf1_out,
		float *help_g,
		float scalefactor
)
{
	// ### Implement me ###
}


__global__ void addKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y*pitchf1 + x;
	if (x < nx && y < ny) {
		accumulator_g[idx] += increment_g[idx];
	}
}

__global__ void subKernel
(
		const float *increment_g,
		float *accumulator_g,
		int   nx,
		int   ny,
		int   pitchf1
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y*pitchf1 + x;
	if (x < nx && y < ny) {
		accumulator_g[idx] -= increment_g[idx];
	}
}

__global__ void setKernel
(
		float *field_g,
		int   nx,
		int   ny,
		int   pitchf1,
		float value
)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int idx = y*pitchf1 + x;
	if (x < nx && y < ny) {
		field_g[idx] = value;
	}
}

